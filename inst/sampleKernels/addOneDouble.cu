
#include <hip/hip_runtime.h>
extern "C"
__global__ void addOne(double* vals, int N)
{
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

    int idx = myblock * blocksize + subthread;

    if(idx < N) {
      vals[idx] = vals[idx] + 1.0;
    }
}

extern "C"
__global__ void addOneFloat(double* vals, int N, float *out)
{
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

    int idx = myblock * blocksize + subthread;

    if(idx < N) {
       out[idx] = (float) vals[idx] + 1.0;
    }
}
