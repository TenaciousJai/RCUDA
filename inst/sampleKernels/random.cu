#include "hip/hip_runtime.h"
extern "C"
{

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel(hiprandState  *state)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;
    hiprand_init(9131, idx, 0, &state[idx]);
    return;
}

__global__ void runif_kernel(hiprandState *state, float *vals, int n, float lo, float hi)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

    if (idx < n){
        vals[idx] = lo + (hi-lo)*hiprand_uniform(&state[idx]);
    }
    return;
}

__global__ void rnorm_kernel(hiprandState *state, float *vals, int n, float mu, float sigma)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

	if (idx < n) {
	    vals[idx] = mu + sigma * hiprand_normal(&state[idx]);
	}
    return;
}

__global__ void rpois_kernel(hiprandState *state, int *vals, int n, double lambda)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

	if (idx < n) {
	    vals[idx] = hiprand_poisson(&state[idx], lambda);
	}
    return;
}

/*
   Few others are available for pseudo-RNGs:

   hiprand_log_normal
   hiprand_uniform_double (double precision uniforms)
   hiprand_normal_double (double precision normals)
   hiprand_log_normal_double (double precision log-normals)
   hiprand_normal2 (pair of float normals)
   hiprand_log_normal2 (pair of float log-normals)
   hiprand_normal2_double (pair of double precision normals)
   hiprand_log_normal2_double (pair of double precision log-normals)
*/


} // END extern 

