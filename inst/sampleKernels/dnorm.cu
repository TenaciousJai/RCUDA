
#include <hip/hip_runtime.h>
extern "C"
__global__ void dnorm_kernel(float *vals, int N, float mu, float sigma)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N) {
            float std = (vals[idx] - mu)/sigma;
	    float e = exp( - 0.5 * std * std);
	    vals[idx] = e / ( sigma * sqrt(2 * 3.141592653589793));
	}
}
