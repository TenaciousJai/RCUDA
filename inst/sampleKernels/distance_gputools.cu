
#include <hip/hip_runtime.h>
#define NUM_THREADS 32
#define size_t int

extern "C"
__global__ void 
euclidean_kernel(const float * vg_a, size_t pitch_a, size_t n_a,
		 const float * vg_b, size_t pitch_b, size_t n_b,
		 size_t k,
		 float * d, size_t pitch_d)
{
  size_t x = blockIdx.x;
  size_t y = blockIdx.y;

  // If an element is to be computed
  if(x < n_a && y < n_b) {

    __shared__ float temp[NUM_THREADS];

    temp[threadIdx.x] = 0.0;
    for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
      float t = vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset];
      temp[threadIdx.x] += (t * t);
    }
    
    // Sync with other threads
    __syncthreads();
    
    // Reduce
    for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
      if(threadIdx.x < stride) {
	temp[threadIdx.x] += temp[threadIdx.x + stride];
      }
      __syncthreads();
    }
    // Write to global memory
    if(threadIdx.x == 0) {
      d[y * pitch_d + x] = sqrt(temp[0]);
    }
  }
}

extern "C"
__global__ void 
euclidean_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a,
		      const float * vg_b, size_t pitch_b, size_t n_b,
		      size_t k, float * d, size_t pitch_d, float p)
{
	size_t x = blockIdx.x, y = blockIdx.y;

	if((x == y) && (x < n_a) && (threadIdx.x == 0))
		d[y * pitch_d + x] = 0.0;
  
	// If all element is to be computed
	if(y < n_a && x < y) {
		__shared__ float temp[NUM_THREADS];    

		temp[threadIdx.x] = 0.0;
    
		for(size_t offset = threadIdx.x; offset < k; offset += NUM_THREADS) {
			float t = vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset];
			temp[threadIdx.x] += (t * t);
		}
    
		// Sync with other threads
		__syncthreads();
    
                // http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html
		// Reduce
		// This is a cumsum.
		// Vital that __syncthreads is called so all threads
                // update temp[threadIdx.x]
		for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
			if(threadIdx.x < stride)
				temp[threadIdx.x] += temp[threadIdx.x + stride];
			__syncthreads();
		}
	    
		// Write to global memory
		if(threadIdx.x == 0) {
			float s = sqrt(temp[0]);
			d[y * pitch_d + x] = s;
			d[x * pitch_d + y] = s;
		}
	}
}




extern "C"
__global__ void minkowski_kernel(const float * vg_a, size_t pitch_a, size_t n_a,
				 const float * vg_b, size_t pitch_b, size_t n_b,
				 size_t k,
				 float * d, size_t pitch_d,
				 float p)
{
	size_t 
		x = blockIdx.x, y = blockIdx.y;

  // If all element is to be computed
  if(x < n_a && y < n_b) {
    __shared__ float temp[NUM_THREADS];

    temp[threadIdx.x] = 0.0;
    for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
      float t = fabsf(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
      temp[threadIdx.x] += __powf(t, p);
    }
    
    // Sync with other threads
    __syncthreads();
    
    // Reduce
		for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
			if(threadIdx.x < stride)
				temp[threadIdx.x] += temp[threadIdx.x + stride];
			__syncthreads();
		}
    // Write to global memory
		if(threadIdx.x == 0) {
			float power = 1.f/p;
			d[y * pitch_d + x] = __powf(temp[0], power);
		}
  }
}


extern "C"
__global__ void canberra_kernel(const float * vg_a, size_t pitch_a, size_t n_a,
				const float * vg_b, size_t pitch_b, size_t n_b,
				size_t k,
				float * d, size_t pitch_d,
				float p)
{
  size_t x = blockIdx.x;
  size_t y = blockIdx.y;

  // If all element is to be computed
  if(x < n_a && y < n_b) {
    __shared__ float temp[NUM_THREADS];

    temp[threadIdx.x] = 0.0;
    for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
      float num = abs(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
      float den = abs(vg_a[x * pitch_a + offset] + vg_b[y * pitch_b + offset]);
      if(den != 0.0) {
	temp[threadIdx.x] += num / den;
      }
    }
    
    // Sync with other threads
    __syncthreads();
    
    // Reduce
    for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
      if(threadIdx.x < stride) {
	temp[threadIdx.x] += temp[threadIdx.x + stride];
      }
      __syncthreads();
    }
    // Write to global memory
    if(threadIdx.x == 0) {
      d[y * pitch_d + x] = temp[0];
    }
  }
}
