
#include <hip/hip_runtime.h>
/* Taken from gputools with the purpos of showing how we can .   */

#define NUM_THREADS 32

/* 
  vg_a and vg_b are two matrices.
  n_a, n_b are the number of rows/observations in the respective matrices.
  pitch_a, pitch_b are  the number of bytes (not elements) between observations in a row, i.e. the stride
  k  - number of variables in each observation, i.e. ncol for each of vg_a and vg_b.
  d - the storage for the answers
  pitch_d - the stride for d giving the offset between elements
  p - ignored by this metric.

  The kernel calls are arranged in a grid of n_a x n_b. So the kernel looks at the block indices
  and only bothers to compute the lower diagonal block of the result matrix.
  It checks if we are on the diagonal (x==y) and if we are in the first thread of the block and sets the result to 0.
  If we are not on the diagonal and x < y, 
 */
extern "C"
__global__ void euclidean_kernel_same(const float * vg_a, size_t pitch_a, 
	size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b,
	size_t k, float * d, size_t pitch_d, float p)
{
	size_t 
		x = blockIdx.x, y = blockIdx.y;

	if((x == y) && (x < n_a) && (threadIdx.x == 0))
		d[y * pitch_d + x] = 0.0;
  
	// If all element is to be computed
	if(y < n_a && x < y) {
		__shared__ float temp[NUM_THREADS];    

		temp[threadIdx.x] = 0.0;
    
		for(size_t offset = threadIdx.x; offset < k; offset += NUM_THREADS) {
			float t = vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset];
			temp[threadIdx.x] += (t * t);
		}
    
		// Sync with other threads
		__syncthreads();
    
		// Reduce
		for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
			if(threadIdx.x < stride)
				temp[threadIdx.x] += temp[threadIdx.x + stride];
			__syncthreads();
		}
	    
		// Write to global memory
		if(threadIdx.x == 0) {
			float s = sqrt(temp[0]);
			d[y * pitch_d + x] = s;
			d[x * pitch_d + y] = s;
		}
	}
}
