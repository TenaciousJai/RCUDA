#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Random Gamma variates in CUDA...
// Surprisingly, there doesn't seem to be any standard way to 
// generate these from the SDK. This is the CUDA port of the 
// rgamma code as used by R.

extern "C"
{

// __constants__ for speed:
/*
__const__ float sqrt32 = 5.656854;
__const__ float exp_m1 = 0.36787944117144232159;
__const__ float q1 = 0.04166669;
__const__ float q2 = 0.02083148;
__const__ float q3 = 0.00801191;
__const__ float q4 = 0.00144121;
__const__ float q5 = -7.388e-5;
__const__ float q6 = 2.4511e-4;
__const__ float q7 = 2.424e-4;
__const__ float a1 = 0.3333333;
__const__ float a2 = -0.250003;
__const__ float a3 = 0.2000062;
__const__ float a4 = -0.1662921;
__const__ float a5 = 0.1423657;
__const__ float a6 = -0.1367177;
__const__ float a7 = 0.1233795;


*/

__global__ void setup_kernel(hiprandState  *state, int *seeds, int n)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;
    if (idx < n){
        hiprand_init(seeds[idx], idx, 0, &state[idx]);
    }
    return;
}

__global__ void rgamma_kernel(hiprandState *state, float *vals, int n, float a, float scale)
{
    float sqrt32 = 5.656854;
    float exp_m1 = 0.36787944117144232159;
    float q1 = 0.04166669;
    float q2 = 0.02083148;
    float q3 = 0.00801191;
    float q4 = 0.00144121;
    float q5 = -7.388e-5;
    float q6 = 2.4511e-4;
    float q7 = 2.424e-4;
    float a1 = 0.3333333;
    float a2 = -0.250003;
    float a3 = 0.2000062;
    float a4 = -0.1662921;
    float a5 = 0.1423657;
    float a6 = -0.1367177;
    float a7 = 0.1233795;

    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

	if (idx >= n) {
	    return;
	}

    // State variables:
    float aa = 0.;
    float aaa = 0.;
    float s, s2, d;    
    float q0, b, si, c;
    float e, p, q, r, t, u, v, w, x, ret_val;

    // Note: Error checks on arguments removed!
    if (a < 1.0) { // GS algorithm for parameters a < 1
        if (a == 0){
            vals[idx] = 0.0;
	        return;
        }
        e = 1.0 + exp_m1 * a;
        while (1){
            p = e * hiprand_normal(&state[idx]);
            if (p >= 1.0) {
                x = -logf((e - p)/a);
                if (logf(hiprand_uniform(&state[idx])) <= (1.0 - a) * logf(x)){
                    break;
                }
            } else {
                x = expf(logf(p) / a);
                if (logf(hiprand_uniform(&state[idx])) <= x){
                    break;
                }
            }
        }
        vals[idx] = scale * x;
        return;
    }

    // --- a >= 1 : GD algorithm --- 

    // Step 1: Recalculations of s2, s, d if a has changed 
    if (a != aa) {
        aa = a;
        s2 = a - 0.5;
        s = sqrtf(s2);
        d = sqrt32 - s * 12.0;
    }

    // Step 2: t = standard normal deviate,
    //           x = (s,1/2) -normal deviate. 

    //printf("Step 2...\n");

    // immediate acceptance (i)
    t = hiprand_normal(&state[idx]);
    x = s + 0.5 * t;
    ret_val = x * x;
    if (t >= 0.0){
        vals[idx] = scale * ret_val;
        return;
    }

    //printf("Step 3...\n");

    // Step 3: u = 0,1 - uniform sample. squeeze acceptance (s)
    u = hiprand_uniform(&state[idx]);
    if ((d*u) <= (t*t*t)){
	    vals[idx] = scale * ret_val;
        return;
    }

    //printf("Step 4...\n");

    // Step 4: recalculations of q0, b, si, c if necessary
    if (a != aaa){
        aaa = a;
        r = 1.0 / a;
        q0 = ((((((q7 * r + q6) * r + q5) * r + q4) * r + q3) * r 
                    + q2) * r + q1) * r;

        // Approximation depending on size of parameter a 
	    // The constants in the expressions for b, si and c 
	    // were established by numerical experiments 

	    if (a <= 3.686) {
	        b = 0.463 + s + 0.178 * s2;
	        si = 1.235;
	        c = 0.195 / s - 0.079 + 0.16 * s;
	    } else if (a <= 13.022) {
	        b = 1.654 + 0.0076 * s2;
	        si = 1.68 / s + 0.275;
	        c = 0.062 / s + 0.024;
	    } else {
	        b = 1.77;
	        si = 0.75;
	        c = 0.1515 / s;
	    }

    }

    //printf("Step 5...\n");

    // Step 5: no quotient test if x not positive 
    if (x > 0.0) {
    	
        // Step 6: calculation of v and quotient q 
	    v = t / (s + s);
	    if (fabs(v) <= 0.25){
	        q = q0 + 0.5 * t * t * ((((((a7 * v + a6) * v + a5) * v + a4) * v
				      + a3) * v + a2) * v + a1) * v;
	    } else {
	        q = q0 - s * t + 0.25 * t * t + (s2 + s2) * log(1.0 + v);
        }

	    // Step 7: quotient acceptance (q) 
	    if (logf(1.0 - u) <= q){
	        vals[idx] = scale * ret_val;
            return;
        }

    }

    //printf("Step 8...\n");

    while (1){

	    // Step 8: e = standard exponential deviate
	    //	u =  0,1 -uniform deviate
	    //	t = (b,si)-double exponential (laplace) sample
	    e = -logf(hiprand_uniform(&state[idx])); // Expo(1)
	    u = hiprand_uniform(&state[idx]);
    	u = u + u - 1.0;
	    if (u < 0.0){
	        t = b - si * e;
        } else {
	        t = b + si * e;
        }
    	// Step	 9:  rejection if t < tau(1) = -0.71874483771719
	    if (t >= -0.71874483771719) {
	        // Step 10:	 calculation of v and quotient q 
    	    v = t / (s + s);
	        if (fabs(v) <= 0.25){
		        q = q0 + 0.5 * t * t *
		            ((((((a7 * v + a6) * v + a5) * v + a4) * v + a3) * v
		            + a2) * v + a1) * v;
            } else {
		        q = q0 - s * t + 0.25 * t * t + (s2 + s2) * log(1.0 + v);
            }
	        // Step 11:	 hat acceptance (h) 
	        // (if q not positive go to step 8) 
	        if (q > 0.0) {
		        w = expm1f(q); // expm1 is double prec...
		        //  ^^^^^ original code had approximation with rel.err < 2e-7 
		        // if t is rejected sample again at step 8 
		        if (c * fabs(u) <= w * expf(e - 0.5 * t * t)){
		            break;
                }
	        }
	    }

    } // repeat .. until  `t' is accepted 

    //printf("Returning at end...\n");

    x = s + 0.5 * t;
    vals[idx] = scale * x * x;
    return;
}

}


/*
   // R CODE from rgamma.c

#include "nmath.h"

#define repeat for(;;)

double rgamma(double a, double scale)
{
    // Constants:
    const static double sqrt32 = 5.656854;
    const static double exp_m1 = 0.36787944117144232159; // exp(-1) = 1/e 

    // Coefficients q[k] - for q0 = sum(q[k]*a^(-k))
    // Coefficients a[k] - for q = q0+(t*t/2)*sum(a[k]*v^k)
    // Coefficients e[k] - for exp(q)-1 = sum(e[k]*q^k)
    
    const static double q1 = 0.04166669;
    const static double q2 = 0.02083148;
    const static double q3 = 0.00801191;
    const static double q4 = 0.00144121;
    const static double q5 = -7.388e-5;
    const static double q6 = 2.4511e-4;
    const static double q7 = 2.424e-4;

    const static double a1 = 0.3333333;
    const static double a2 = -0.250003;
    const static double a3 = 0.2000062;
    const static double a4 = -0.1662921;
    const static double a5 = 0.1423657;
    const static double a6 = -0.1367177;
    const static double a7 = 0.1233795;

    // State variables [FIXME for threading!] :
    static double aa = 0.;
    static double aaa = 0.;
    static double s, s2, d;    // no. 1 (step 1)
    static double q0, b, si, c;// no. 2 (step 4)

    double e, p, q, r, t, u, v, w, x, ret_val;

    if (!R_FINITE(a) || !R_FINITE(scale) || a < 0.0 || scale <= 0.0) {
	if(scale == 0.) return 0.;
	ML_ERR_return_NAN;
    }

    if (a < 1.) { // GS algorithm for parameters a < 1
	if(a == 0)
	    return 0.;
	e = 1.0 + exp_m1 * a;
	repeat {
	    p = e * unif_rand();
	    if (p >= 1.0) {
		x = -log((e - p) / a);
		if (exp_rand() >= (1.0 - a) * log(x))
		    break;
	    } else {
		x = exp(log(p) / a);
		if (exp_rand() >= x)
		    break;
	    }
	}
	return scale * x;
    }

    // --- a >= 1 : GD algorithm --- 

    // Step 1: Recalculations of s2, s, d if a has changed 
    if (a != aa) {
	aa = a;
	s2 = a - 0.5;
	s = sqrt(s2);
	d = sqrt32 - s * 12.0;
    }
    // Step 2: t = standard normal deviate,
    //           x = (s,1/2) -normal deviate. 

    // immediate acceptance (i)
    t = norm_rand();
    x = s + 0.5 * t;
    ret_val = x * x;
    if (t >= 0.0)
	return scale * ret_val;

    // Step 3: u = 0,1 - uniform sample. squeeze acceptance (s)
    u = unif_rand();
    if (d * u <= t * t * t)
	return scale * ret_val;

    // Step 4: recalculations of q0, b, si, c if necessary

    if (a != aaa) {
	aaa = a;
	r = 1.0 / a;
	q0 = ((((((q7 * r + q6) * r + q5) * r + q4) * r + q3) * r
	       + q2) * r + q1) * r;

	// Approximation depending on size of parameter a 
	// The constants in the expressions for b, si and c 
	// were established by numerical experiments 

	if (a <= 3.686) {
	    b = 0.463 + s + 0.178 * s2;
	    si = 1.235;
	    c = 0.195 / s - 0.079 + 0.16 * s;
	} else if (a <= 13.022) {
	    b = 1.654 + 0.0076 * s2;
	    si = 1.68 / s + 0.275;
	    c = 0.062 / s + 0.024;
	} else {
	    b = 1.77;
	    si = 0.75;
	    c = 0.1515 / s;
	}
    }
    // Step 5: no quotient test if x not positive 

    if (x > 0.0) {
	// Step 6: calculation of v and quotient q 
	v = t / (s + s);
	if (fabs(v) <= 0.25)
	    q = q0 + 0.5 * t * t * ((((((a7 * v + a6) * v + a5) * v + a4) * v
				      + a3) * v + a2) * v + a1) * v;
	else
	    q = q0 - s * t + 0.25 * t * t + (s2 + s2) * log(1.0 + v);


	// Step 7: quotient acceptance (q) 
	if (log(1.0 - u) <= q)
	    return scale * ret_val;
    }

    repeat {
	// Step 8: e = standard exponential deviate
	//	u =  0,1 -uniform deviate
	//	t = (b,si)-double exponential (laplace) sample
	e = exp_rand();
	u = unif_rand();
	u = u + u - 1.0;
	if (u < 0.0)
	    t = b - si * e;
	else
	    t = b + si * e;
	// Step	 9:  rejection if t < tau(1) = -0.71874483771719
	if (t >= -0.71874483771719) {
	    // Step 10:	 calculation of v and quotient q 
	    v = t / (s + s);
	    if (fabs(v) <= 0.25)
		q = q0 + 0.5 * t * t *
		    ((((((a7 * v + a6) * v + a5) * v + a4) * v + a3) * v
		      + a2) * v + a1) * v;
	    else
		q = q0 - s * t + 0.25 * t * t + (s2 + s2) * log(1.0 + v);
	    // Step 11:	 hat acceptance (h) 
	    // (if q not positive go to step 8) 
	    if (q > 0.0) {
		w = expm1(q);
		//  ^^^^^ original code had approximation with rel.err < 2e-7 
		// if t is rejected sample again at step 8 
		if (c * fabs(u) <= w * exp(e - 0.5 * t * t))
		    break;
	    }
	}
    } // repeat .. until  `t' is accepted 
    x = s + 0.5 * t;
    return scale * x * x;
}

*/

