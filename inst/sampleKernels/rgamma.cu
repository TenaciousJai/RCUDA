/*
 *  rgamma.cu : GPU-based generation of Gamma random variates, 
 *  based on rgamma.c. 
 *
 *  Full license for rgamma.c below...
 *
 *  Mathlib : A C Library of Special Functions
 *  Copyright (C) 1998 Ross Ihaka
 *  Copyright (C) 2000--2008 The R Core Team
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program; if not, a copy is available at
 *  http://www.r-project.org/Licenses/
 *
 *  SYNOPSIS
 *
 *    #include <Rmath.h>
 *    double rgamma(double a, double scale);
 *
 *  DESCRIPTION
 *
 *    Random variates from the gamma distribution.
 *
 *  REFERENCES
 *
 *    [1] Shape parameter a >= 1.  Algorithm GD in:
 *
 *	  Ahrens, J.H. and Dieter, U. (1982).
 *	  Generating gamma variates by a modified
 *	  rejection technique.
 *	  Comm. ACM, 25, 47-54.
 *
 *
 *    [2] Shape parameter 0 < a < 1. Algorithm GS in:
 *
 *	  Ahrens, J.H. and Dieter, U. (1974).
 *	  Computer methods for sampling from gamma, beta,
 *	  poisson and binomial distributions.
 *	  Computing, 12, 223-246.
 *
 *    Input: a = parameter (mean) of the standard gamma distribution.
 *    Output: a variate from the gamma(a)-distribution
 */


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Random Gamma variates in CUDA...
// Surprisingly, there doesn't seem to be any standard way to 
// generate these from the SDK. This is the CUDA port of the 
// rgamma code as used by R.

extern "C"
{

// __constants__ (actually slower than keeping within kernel):
/*
__const__ float sqrt32 = 5.656854;
__const__ float exp_m1 = 0.36787944117144232159;
__const__ float q1 = 0.04166669;
__const__ float q2 = 0.02083148;
__const__ float q3 = 0.00801191;
__const__ float q4 = 0.00144121;
__const__ float q5 = -7.388e-5;
__const__ float q6 = 2.4511e-4;
__const__ float q7 = 2.424e-4;
__const__ float a1 = 0.3333333;
__const__ float a2 = -0.250003;
__const__ float a3 = 0.2000062;
__const__ float a4 = -0.1662921;
__const__ float a5 = 0.1423657;
__const__ float a6 = -0.1367177;
__const__ float a7 = 0.1233795;
*/


__global__ void setup_kernel(hiprandState  *state, int *seeds, int n)
{
    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;
    if (idx < n){
        hiprand_init(seeds[idx], idx, 0, &state[idx]);
    }
    return;
}

__global__ void rgamma_kernel(hiprandState *state, int state_len, float *vals, int n, float a, float scale)
{
    float sqrt32 = 5.656854;
    float exp_m1 = 0.36787944117144232159;
    float q1 = 0.04166669;
    float q2 = 0.02083148;
    float q3 = 0.00801191;
    float q4 = 0.00144121;
    float q5 = -7.388e-5;
    float q6 = 2.4511e-4;
    float q7 = 2.424e-4;
    float a1 = 0.3333333;
    float a2 = -0.250003;
    float a3 = 0.2000062;
    float a4 = -0.1662921;
    float a5 = 0.1423657;
    float a6 = -0.1367177;
    float a7 = 0.1233795;

    // 
    // INPUTS
    // ======
    //
    // state        :: RNG states (vector of length state_len)
    //
    // state_len    :: length of rng states (one thread per rng_state)
    //                 if (idx >= state_len){ do nothing... }
    //
    // vals         :: storage for the n samples to be generated
    //
    // n            :: length of vector to sample
    //
    // a            :: alpha
    //
    // scale        :: (1/beta) [possibly -- check, may be beta]
    //
    // ===================================================================
    //
    // From these we derive:
    //
    // n_per_thread :: number of rng's to generate per thread
    //                 heurestically, each thread computes
    //                 n_per_thread = n / state_len 
    //                 variates. However, if n_per_thread is
    //                 not an integer, then some do one less,
    //                 some one more.
    //
    // n(idx)       :: number of rng's that thread idx will generate
    //
    // n_lo         :: minimum number of rng's that a (used) thread
    //                 will generate
    //
    // n_hi         :: maximum number of rng's that a (used) thread
    //                 will generate
    //
    // n_extra      :: (= n % state_len) the number of threads that
    //                 will generate n_hi rng's
    //
    // n_regular    :: (= state_len - n_extra) the number of threads that
    //                 will generate n_lo rng's
    // 
    // Note:
    //
    // n = (n_lo * n_regular) + (n_hi * n_extra)
    //
    // e.g.,
    //
    // n = 108, state_len = 40
    // => n_lo = 2, n_hi = 3
    // => n_extra = 28, n_regular = 12
    // => threads  1-28 generate 3 rng's
    // => threads 29-40 generate 2 rngs'
    // 
    // thread idx writes to:
    // 
    // if (idx < n_extra){
    //   // 'extra' thread:
    //   n_hi*idx : ((n_hi*idx) + (n_hi-1))
    // }
    // if (idx >= n_extra){
    //   // 'regular' thread:
    //   (n_hi*n_extra) + (n_lo*(idx-n_extra)):((n_lo*(idx-n_extra)) + (n_lo-1))
    // }
    // 
    // e.g., (cont...)
    //
    // thread  0 writes to: 0,1,2
    // thread  1 writes to: 3,4,5
    // ...
    // thread 27 writes to: 81,82,83
    // thread 28 writes to: 84,85
    // thread 29 writes to: 86,87
    // ...
    // thread 40 writes to: 118,119
    //

    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;

	if (idx >= state_len) {
	    return;
	}

    // Number of regular and extra threads:
    int n_extra = (n % state_len);
    //int n_regular = n - n_extra;
    int n_lo = (int) (n/state_len);
    int n_hi = (n_lo+1);
    int out_idx_start, n_per_thread;
    if (idx < n_extra){
        n_per_thread = n_hi;
        out_idx_start = n_hi*idx;
    } else {
        n_per_thread = n_lo;
        out_idx_start = n_hi*n_extra + n_lo*(idx-n_extra);
    }
    int gen_num;

    // State variables:
    float aa = 0.0;
    float aaa = 0.0;
    float s, s2, d;    
    float q0, b, si, c;
    float e, p, q, r, t, u, v, w, x, ret_val;

    // Generate RNG's for this thread...
    for (gen_num=0; gen_num<n_per_thread; gen_num++){

        // Note: Error checks on arguments removed!
        if (a < 1.0) { // GS algorithm for parameters a < 1
            if (a == 0){
                vals[out_idx_start+gen_num] = 0.0;
	            continue;
            }
            e = 1.0 + exp_m1 * a;
            while (1){
                p = e * hiprand_normal(&state[idx]);
                if (p >= 1.0) {
                    x = -logf((e - p)/a);
                    if (logf(hiprand_uniform(&state[idx])) <= (1.0 - a) * logf(x)){
                        break;
                    }
                } else {
                    x = expf(logf(p) / a);
                    if (logf(hiprand_uniform(&state[idx])) <= x){
                        break;
                    }
                }
            }
            vals[out_idx_start+gen_num] = scale * x;
            continue;
        }

        // --- a >= 1 : GD algorithm --- 

        // Step 1: Recalculations of s2, s, d if a has changed 
        if (a != aa) {
            aa = a;
            s2 = a - 0.5;
            s = sqrtf(s2);
            d = sqrt32 - s * 12.0;
        }

        // Step 2: t = standard normal deviate,
        //           x = (s,1/2) -normal deviate. 

        //printf("Step 2...\n");

        // immediate acceptance (i)
        t = hiprand_normal(&state[idx]);
        x = s + 0.5 * t;
        ret_val = x * x;
        if (t >= 0.0){
            vals[out_idx_start+gen_num] = scale * ret_val;
            continue;
        }

        //printf("Step 3...\n");

        // Step 3: u = 0,1 - uniform sample. squeeze acceptance (s)
        u = hiprand_uniform(&state[idx]);
        if ((d*u) <= (t*t*t)){
	        vals[out_idx_start+gen_num] = scale * ret_val;
            continue;
        }

        //printf("Step 4...\n");

        // Step 4: recalculations of q0, b, si, c if necessary
        if (a != aaa){
            aaa = a;
            r = 1.0 / a;
            q0 = ((((((q7 * r + q6) * r + q5) * r + q4) * r + q3) * r 
                    + q2) * r + q1) * r;

            // Approximation depending on size of parameter a 
	        // The constants in the expressions for b, si and c 
	        // were established by numerical experiments 

    	    if (a <= 3.686) {
	            b = 0.463 + s + 0.178 * s2;
	            si = 1.235;
	            c = 0.195 / s - 0.079 + 0.16 * s;
    	    } else if (a <= 13.022) {
	            b = 1.654 + 0.0076 * s2;
	            si = 1.68 / s + 0.275;
	            c = 0.062 / s + 0.024;
    	    } else {
	            b = 1.77;
	            si = 0.75;
	            c = 0.1515 / s;
	        }

        }

        //printf("Step 5...\n");

        // Step 5: no quotient test if x not positive 
        if (x > 0.0) {
    	
            // Step 6: calculation of v and quotient q 
	        v = t / (s + s);
    	    if (fabs(v) <= 0.25){
	            q = q0 + 0.5 * t * t * ((((((a7 * v + a6) * v + a5) * v + a4) * v
		    		      + a3) * v + a2) * v + a1) * v;
	        } else {
	            q = q0 - s * t + 0.25 * t * t + (s2 + s2) * log(1.0 + v);
            }

    	    // Step 7: quotient acceptance (q) 
	        if (logf(1.0 - u) <= q){
	            vals[out_idx_start+gen_num] = scale * ret_val;
                continue;
            }

        }

        //printf("Step 8...\n");

        while (1){

	        // Step 8: e = standard exponential deviate
	        //	u =  0,1 -uniform deviate
	        //	t = (b,si)-double exponential (laplace) sample
    	    e = -logf(hiprand_uniform(&state[idx])); // Expo(1)
	        u = hiprand_uniform(&state[idx]);
    	    u = u + u - 1.0;
    	    if (u < 0.0){
	            t = b - si * e;
            } else {
	            t = b + si * e;
            }
    	    // Step	 9:  rejection if t < tau(1) = -0.71874483771719
    	    if (t >= -0.71874483771719) {
	            // Step 10:	 calculation of v and quotient q 
    	        v = t / (s + s);
	            if (fabs(v) <= 0.25){
		            q = q0 + 0.5 * t * t *
		                ((((((a7 * v + a6) * v + a5) * v + a4) * v + a3) * v
		                + a2) * v + a1) * v;
                } else {
	    	        q = q0 - s * t + 0.25 * t * t + (s2 + s2) * log(1.0 + v);
                }
	            // Step 11:	 hat acceptance (h) 
    	        // (if q not positive go to step 8) 
	            if (q > 0.0) {
		            w = expm1f(q); // expm1 is double prec...
		            //  ^^^^^ original code had approximation with rel.err < 2e-7 
		            // if t is rejected sample again at step 8 
    		        if (c * fabs(u) <= w * expf(e - 0.5 * t * t)){
	    	            break;
                    }
	            }
	        }

        } // repeat .. until  `t' is accepted 

        //printf("Returning at end...\n");

        x = s + 0.5 * t;
        vals[out_idx_start+gen_num] = scale * x * x;
        continue;
    
    } // end gen_num loop

    return;
}

}

/*
   // Full C wrapper for debugging...

int main()
{
    int n = 100000;
    int n_states = 1000;
    int threads_per_block = 512;
    
    int h_seeds[n_states];
    int i;
    for (i=0; i<n_states; i++){
        h_seeds[i] = 198+6*i;
    }
    int *d_seeds;
    cudaMalloc((void **)&d_seeds, n_states*sizeof(int));
    cudaMemcpy(d_seeds,h_seeds,n_states*sizeof(int),cudaMemcpyHostToDevice);

    curandState *state;
    cudaMalloc((void**)&state, n_states*sizeof(curandState)); 

    dim3 block_dims(threads_per_block,1,1); 
    int grid_d1 = (int)floor(sqrt((float)n_states/(float)threads_per_block));
    int grid_d2 = (int)ceil((float)n_states/(float)(grid_d1*threads_per_block));
    dim3 grid_dims(grid_d1, grid_d2, 1);

    printf("Calling setup kernel...\n");

    setup_kernel<<<grid_dims,block_dims>>>(state,d_seeds,n_states);

    printf("done with setup kernel. Calling rgamma kernel...\n");

    float a = 10.0;
    float b = 1.2;
    float oob = 1.0/b;
    float h_vals[n];
    for (i=0; i<n; i++){
        h_vals[i] = 0.0;
    }
    float *d_vals;
    cudaMalloc((void**)&d_vals,n*sizeof(float));
    cudaMemcpy(d_vals,h_vals,n_states*sizeof(int),cudaMemcpyHostToDevice);
    
    rgamma_kernel<<<grid_dims,block_dims>>>(state,n_states,d_vals,n,a,oob);

    printf("done with call. Copying results back...\n");

    cudaMemcpy(h_vals,d_vals,n*sizeof(int),cudaMemcpyDeviceToHost);
    
    cudaDeviceSynchronize();
    
    printf("done.\n");

    printf("output:\n[1]");
    for (i=0; i<n; i++){
        printf("   %4.3f",h_vals[i]);
        if ((i+1)%8 == 0)
            printf("\n[%d]",i+2);
    }
    printf("\n");

    return 0;
}

*/

