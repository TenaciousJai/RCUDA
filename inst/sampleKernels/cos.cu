
#include <hip/hip_runtime.h>
/* Taken from CUDA_example.cu from Paul Baine's talk on GPUs. 
Originally taken from some other site, I believe.
*/


#define COS_THREAD_CNT 512
#define N 10000000
#define TWO_PI 6.283185

/* --------------------------- target code ------------------------------*/
struct cosParams {
        float *arg;
        float *res;
        int n;
};

extern "C"
__global__ void cos_main(struct cosParams params)
{
        // Computes:
        // res[threadIdx.x + 0*COS_THREAD_CNT] = cos(threadIdx.x + 0*COS_THREAD_CNT)
	// res[threadIdx.x + 1*COS_THREAD_CNT] = cos(threadIdx.x + 1*COS_THREAD_CNT)
        // res[threadIdx.x + 2*COS_THREAD_CNT] = cos(threadIdx.x + 2*COS_THREAD_CNT)
        // ...etc...
	//
	// if COS_THREAD_
	// if COS_THREAD_CNT = 1, this computes all of the cosines in one go,
	// otherwise, it spreads it out across threads...

	int i;
	for (i=threadIdx.x; i<params.n; i+=COS_THREAD_CNT){
		params.res[i] = __cosf(params.arg[i]);
	}
	return;
}

