
#include <hip/hip_runtime.h>
extern "C"
__global__ void euclideanDistance(const float *data_a, int nrow_a, const float *data_b, int nrow_b, int ncol, float *ans)
{
/*
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
*/

    /* This is very simple, limited to 512 pairs of observations in total. */
    int i = threadIdx.x, j = threadIdx.y;

    if(i < nrow_a && j < nrow_b) {
	int off_a = i,
	    off_b = j;
	float sum = 0;
	for(int k = 0; k < ncol ; k++, off_a += nrow_a, off_b+= nrow_b) {
	    float tmp = (data_a[off_a] - data_b[off_b]);
	    sum += tmp * tmp;
	}
	ans[i + j*nrow_a] = sqrt(sum);
    }

}


#if 0
/* Taken from gputools package and adapted very slightly to generalize the NUM_THREADS to make 
   this dynamic via the blockDim.x */

__global__ void euclideanReduceKernel(const float * vg_a, size_t pitch_a, 
	size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b,
	size_t k, float * d, size_t pitch_d, float p)
{
	size_t x = blockIdx.x, y = blockIdx.y;

	size_t numThreadsPerPair = blockDim.x;

	if((x == y) && (x < n_a) && (threadIdx.x == 0))
		d[y * pitch_d + x] = 0.0;
  
	// If all element is to be computed
	if(y < n_a && x < y) {
		__shared__ float temp[numThreadsPerPair];    

		temp[threadIdx.x] = 0.0;
    
		for(size_t offset = threadIdx.x; offset < k; offset += numThreadsPerPair) {
			float t = vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset];
			temp[threadIdx.x] += (t * t);
		}
    
		// Sync with other threads
		__syncthreads();
    
                // http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html
		// This is a cumsum.
		// Vital that __syncthreads is called so all threads
                // update temp[threadIdx.x]
		// Reduce
		for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
			if(threadIdx.x < stride)
				temp[threadIdx.x] += temp[threadIdx.x + stride];
			__syncthreads();
		}
	    
		// Write to global memory
		if(threadIdx.x == 0) {
			float s = sqrt(temp[0]);
			d[y * pitch_d + x] = s;
			d[x * pitch_d + y] = s;
		}
	}
}
#endif
