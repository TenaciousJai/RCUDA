
#include <hip/hip_runtime.h>
extern "C"
__global__ void dnorm_kernel(float *vals, int N, float mu, float sigma)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N) {
	   vals[idx] = sigma;
	}
}
