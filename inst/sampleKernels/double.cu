
#include <hip/hip_runtime.h>
extern "C"
__global__ void add_kernel(double *vals, double *ans, int N, double mu)
{
       // Taken from geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
           /* how big is each block within a grid */
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
            /* get thread within a block */
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

    int idx = myblock * blocksize + subthread;

    if(idx < N) 
       ans[idx] = vals[idx] + mu;
}


extern "C"
__global__ void fadd_kernel(float *vals, int N, float mu)
{
       // Taken from geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
           /* how big is each block within a grid */
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
            /* get thread within a block */
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

    int idx = myblock * blocksize + subthread;

    if(idx < N) 
       vals[idx] = mu;
}

extern "C"
__global__ void dset_kernel(double *vals, int N, double mu)
{
       // Taken from geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
           /* how big is each block within a grid */
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
            /* get thread within a block */
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

    int idx = myblock * blocksize + subthread;

    if(idx < N) 
       vals[idx] = mu;
}


extern "C"
__global__ void dset_both_kernel(double *vals, int N, double mu, float sd)
{
       // Taken from geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
           /* how big is each block within a grid */
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
            /* get thread within a block */
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

    int idx = myblock * blocksize + subthread;

    if(idx < N) 
       vals[idx] = mu + sd;
}


extern "C"
__global__ void dset_array_kernel(double *vals, int N, double *mu)
{
       // Taken from geco.mines.edu/workshop/aug2010/slides/fri/cuda1.pd
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
           /* how big is each block within a grid */
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
            /* get thread within a block */
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;

    int idx = myblock * blocksize + subthread;

    if(idx < N) 
       vals[idx] = mu[0];
}
