
#include <hip/hip_runtime.h>
extern "C"
__global__ void setValue_kernel(int *vals, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(0 && idx < N) 
	    vals[idx] = idx;

}
