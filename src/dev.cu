#include <hip/hip_runtime.h>
#include <Rdefines.h>

SEXP
R_cudaGetDeviceCount()
{
  int num = 0;
  hipError_t err = hipGetDeviceCount(&num);
  if(err != hipSuccess) {
      PROBLEM "%s", hipGetErrorString(err)
      ERROR;
  }
  return(ScalarString(num));
}
