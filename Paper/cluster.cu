#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<hip/hip_math_constants.h>
//#include<cuseful.h>
//#include <R.h>
#include "hcluster.h"

#define NUM_THREADS 32
#define NUM_BLOCKS 1024

#if 0
// Distance matrix
__device__ float * hcluster_dist_d;

// Number of elements in each cluster
__device__ float * hcluster_count_d;

// Arrays for finding the minimum of each row and column containing the minimum
__device__ float * hcluster_min_val_d;
__device__ size_t * hcluster_min_col_d;

// Arrays telling which cluster merged with which cluster
__device__ int * hcluster_sub_d;
__device__ int * hcluster_sup_d;

// Array of the values merged at
__device__ float * hcluster_merge_val_d;
#endif

__global__ void complete_kernel(float * dist, const size_t pitch_dist, 
	const size_t n, const int * sub, const int * sup, const float * count, 
	const float * val, const size_t iter, const size_t col_offset, 
	const float lambda, const float beta)
{
  const size_t col = col_offset + NUM_THREADS * blockIdx.x + threadIdx.x;

  // If it matters
  if(col < n) {

    int 
		col_winner = sub[iter], row_winner = sup[iter];
    float 
		top_val = dist[col_winner * pitch_dist + col], 
		bot_val = dist[row_winner * pitch_dist + col];

    bot_val = fmaxf(bot_val, top_val);
    if((col == col_winner) || (col == row_winner))
		bot_val = HIP_INF_F;

    top_val = HIP_INF_F;

    // Write out
    dist[col_winner * pitch_dist + col] = top_val;
    dist[col * pitch_dist + col_winner] = top_val;
    dist[row_winner * pitch_dist + col] = bot_val;
    dist[col * pitch_dist + row_winner] = bot_val;
  }
}

__global__ void convert_kernel(float * dist, size_t pitch_dist, size_t n)
{
  for(size_t index = threadIdx.x; index < n; index += NUM_THREADS) {
    dist[index * pitch_dist + index] = HIP_INF_F;
  }
}
